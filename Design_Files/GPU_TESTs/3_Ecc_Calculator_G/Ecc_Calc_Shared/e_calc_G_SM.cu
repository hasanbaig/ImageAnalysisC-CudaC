<<<<<<< HEAD
//GPU implementation of Eccentricity Calculation
//Author: Hasan Baig
//Email: hasan.baig@hotmail.com 
//Web: http://www.hasanbaig.com 

#include "iostream"
#include "conio.h" 
#include "time.h"
#include "../defines.h"
#include "../my_G_functions.h"
#include "../my_G_functions_SM.h"



int main (void)
{
	int *dev_Image, *dev_hor_scan, *dev_ver_scan, *dev_hor_max_obj, *dev_ver_max_obj;
	float *dev_e_obj;
	
	hipEvent_t start, stop; 
	float time;

	hipEventCreate(&start); 
	hipEventCreate(&stop);

	hipMalloc( (void**)&dev_Image, Image_size );
	hipMalloc( (void**)&dev_hor_scan, hor_scan_size);
	hipMalloc( (void**)&dev_ver_scan, ver_scan_size );
	hipMalloc( (void**)&dev_hor_max_obj, hor_max_obj_size );
	hipMalloc( (void**)&dev_ver_max_obj, ver_max_obj_size );
	hipMalloc( (void**)&dev_e_obj, e_obj_size );

	hipMemcpy( dev_Image, Image, Image_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_hor_max_obj, hor_max_obj, hor_max_obj_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_ver_max_obj, ver_max_obj, ver_max_obj_size,hipMemcpyHostToDevice );

	dim3 dimGrid(2, 2);								
	dim3 dimBlock(17,17,1);	
	
	maj_min_scan_G<<<dimGrid, dimBlock>>>(dev_Image, dev_hor_scan, dev_ver_scan, dev_hor_max_obj, dev_ver_max_obj);


	dim3 dimGrid_e(2, 2);								
	dim3 dimBlock_e(1, 1, 1);	


	//hipDeviceSynchronize();
	hipEventRecord( start, 0 );

	e_calc_G_SM<<<dimGrid_e,dimBlock_e>>>(dev_hor_max_obj, dev_ver_max_obj, dev_e_obj);
	
	//hipDeviceSynchronize();
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop ); 

	printf("\nTIME = %f ms\n\n", time);
	hipEventDestroy( start ); 
	hipEventDestroy( stop );

	hipMemcpy( hor_scan, dev_hor_scan, hor_scan_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( ver_scan, dev_ver_scan, ver_scan_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( hor_max_obj, dev_hor_max_obj, hor_max_obj_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( ver_max_obj, dev_ver_max_obj, ver_max_obj_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( e_obj, dev_e_obj, e_obj_size , hipMemcpyDeviceToHost ); 


	printf("\n"); 
	for(i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for(j=0;j<BLOCKS_X_SCAN;j++)
		{
			printf ("e[%d][%d] = %f\n",i,j,e_obj[i][j]);
		}
	}

	system("pause");
	return 0;
=======
//GPU implementation of Eccentricity Calculation
#include "iostream"
#include "conio.h" 
#include "time.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/defines.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/my_G_functions.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/my_G_functions_SM.h"



int main (void)
{
	int *dev_Image, *dev_hor_scan, *dev_ver_scan, *dev_hor_max_obj, *dev_ver_max_obj;
	float *dev_e_obj;
	
	hipEvent_t start, stop; 
	float time;

	hipEventCreate(&start); 
	hipEventCreate(&stop);

	hipMalloc( (void**)&dev_Image, Image_size );
	hipMalloc( (void**)&dev_hor_scan, hor_scan_size);
	hipMalloc( (void**)&dev_ver_scan, ver_scan_size );
	hipMalloc( (void**)&dev_hor_max_obj, hor_max_obj_size );
	hipMalloc( (void**)&dev_ver_max_obj, ver_max_obj_size );
	hipMalloc( (void**)&dev_e_obj, e_obj_size );

	hipMemcpy( dev_Image, Image, Image_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_hor_max_obj, hor_max_obj, hor_max_obj_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_ver_max_obj, ver_max_obj, ver_max_obj_size,hipMemcpyHostToDevice );

	dim3 dimGrid(2, 2);								
	dim3 dimBlock(17,17,1);	
	
	maj_min_scan_G<<<dimGrid, dimBlock>>>(dev_Image, dev_hor_scan, dev_ver_scan, dev_hor_max_obj, dev_ver_max_obj);


	dim3 dimGrid_e(2, 2);								
	dim3 dimBlock_e(1, 1, 1);	


	//hipDeviceSynchronize();
	hipEventRecord( start, 0 );

	e_calc_G_SM<<<dimGrid_e,dimBlock_e>>>(dev_hor_max_obj, dev_ver_max_obj, dev_e_obj);
	
	//hipDeviceSynchronize();
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop ); 

	printf("\nTIME = %f ms\n\n", time);
	hipEventDestroy( start ); 
	hipEventDestroy( stop );

	hipMemcpy( hor_scan, dev_hor_scan, hor_scan_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( ver_scan, dev_ver_scan, ver_scan_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( hor_max_obj, dev_hor_max_obj, hor_max_obj_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( ver_max_obj, dev_ver_max_obj, ver_max_obj_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( e_obj, dev_e_obj, e_obj_size , hipMemcpyDeviceToHost ); 


	printf("\n"); 
	for(i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for(j=0;j<BLOCKS_X_SCAN;j++)
		{
			printf ("e[%d][%d] = %f\n",i,j,e_obj[i][j]);
		}
	}

	system("pause");
	return 0;
>>>>>>> ac3cb0f79f55229281d36df36921a63db1d105b7
}