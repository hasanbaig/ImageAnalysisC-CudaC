// Noise removing through morphological techniques on GPU
#include "iostream"
#include "conio.h" 
#include "time.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/defines.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/my_C_functions.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/my_G_functions_SM.h"
#include "C:/ProgramData/NVIDIA Corporation/NVIDIA GPU Computing SDK 4.0/C/common/inc/cutil.h"

int main (void)
{
	int *Image_In_dev, *Image_Out_dev;
	int Image_size = Rows * Cols * sizeof(int);
	int Image_Out [Rows][Cols];

	//for debugging
	int Image_Erode[Rows][Cols];
	int Temp[Rows][Cols];
	int *Image_Erode_dev;
	int *Temp_dev;

	hipEvent_t start, stop; 
	float time;

	hipEventCreate(&start); 
	hipEventCreate(&stop);

	dim3 dimGrid(BLOCKS_Y_SCAN, BLOCKS_X_SCAN);								
	dim3 dimBlock(Threads_y_Scan, Threads_x_Scan);

	printf("Original Image with Noise\n\n");
	for (i=0;i<Rows;i++)
	{
		for (j=0;j<Cols;j++)
		{
			printf("%d", Noisy_Image[i][j]);
			if(j==(Rows - 1))
				printf("\n");
		}
	}

	hipMalloc( (void**)&Image_In_dev, Image_size );
	hipMalloc( (void**)&Image_Out_dev, Image_size );
	hipMalloc( (void**)&Image_Erode_dev, Image_size );
	hipMalloc( (void**)&Temp_dev, Image_size );

	hipMemcpy( Image_In_dev, Noisy_Image, Image_size,hipMemcpyHostToDevice );
	//hipMemcpy( Image_Out_dev, Image_Out, Image_size,hipMemcpyHostToDevice );
	//hipDeviceSynchronize();
	hipEventRecord( start, 0 );
	im_open_G_SM<<<dimGrid, dimBlock>>>(Image_In_dev, Image_Out_dev, Image_Erode_dev, Temp_dev);	
	//hipDeviceSynchronize();
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop ); 
	hipEventElapsedTime( &time, start, stop ); 

	printf("\nTIME = %f \n\n", time);

//	hipMemcpy( Image_Erode, Image_Erode_dev, Image_size , hipMemcpyDeviceToHost );
//	hipMemcpy( Temp, Temp_dev, Image_size , hipMemcpyDeviceToHost );
	hipMemcpy( Image_Out, Image_Out_dev, Image_size , hipMemcpyDeviceToHost );


	printf("Noise Free Image\n\n");
	for (i=0;i<Rows;i++)
	{
		for (j=0;j<Cols;j++)
		{
			printf("%d", Image_Out[i][j]);
			if(j==(Rows - 1))
				printf("\n");
		}
	}

	system ("pause");
	return 0;
}