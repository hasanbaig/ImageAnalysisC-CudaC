//Integrated 
#include "iostream"
#include "conio.h" 
#include "time.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/defines.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/my_G_functions.h"

int main(void)
{
	int		*Image_In_dev,
			*Image_Out_dev, 
			*Image_Erode_dev, 
			*Temp_dev, 
			*dev_hor_scan,
			*dev_ver_scan, 
			*dev_hor_max_obj, 
			*dev_ver_max_obj;

	float	*dev_e_obj;
	
	int		Image_Out [Rows][Cols];
	int		Image_Erode[Rows][Cols];
	int		Temp[Rows][Cols];
	int		Image_size = Rows * Cols * sizeof(int);

	hipEvent_t start, stop; 
	float time1,time2,time3;

	hipEventCreate(&start); 
	hipEventCreate(&stop);

	printf("Original Image with Noise\n\n");
	for (i=0;i<Rows;i++)
	{
		for (j=0;j<Cols;j++)
		{
			printf("%d", Noisy_Image[i][j]);
			if(j==(Rows - 1))
				printf("\n");
		}
	}

	hipMalloc( (void**)&Image_In_dev, Image_size );
	hipMalloc( (void**)&Image_Out_dev, Image_size );
	hipMalloc( (void**)&Image_Erode_dev, Image_size );
	hipMalloc( (void**)&Temp_dev, Image_size );
	hipMalloc( (void**)&dev_hor_scan, hor_scan_size );
	hipMalloc( (void**)&dev_ver_scan, ver_scan_size );
	hipMalloc( (void**)&dev_hor_max_obj, hor_max_obj_size );
	hipMalloc( (void**)&dev_ver_max_obj, ver_max_obj_size);
	hipMalloc( (void**)&dev_e_obj, e_obj_size );

	
	//************************** Noise Removing Kernel ************************
	
	hipMemcpy( Image_In_dev, Noisy_Image, Image_size,hipMemcpyHostToDevice );

	dim3 dimGrid_1(BLOCKS_X_MORPH, BLOCKS_Y_MORPH);								
	dim3 dimBlock_1(Threads_x_Morph, Threads_y_Morph);

	hipEventRecord( start, 0 );
	im_open_G <<<dimGrid_1, dimBlock_1>>>
		(
			Image_In_dev, 
			Image_Out_dev, 
			Image_Erode_dev, 
			Temp_dev
		);

	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time1, start, stop ); 

	printf("\nTIME to Run Noise Removing Kernel = %f ms\n\n", time1);
	hipEventDestroy( start ); 
	hipEventDestroy( stop );


	hipMemcpy( Image_Out, Image_Out_dev, Image_size , hipMemcpyDeviceToHost );

	printf("Noise Free Image\n\n");
	for (i=0;i<Rows;i++)
	{
		for (j=0;j<Cols;j++)
		{
			printf("%d", Image_Out[i][j]);
			if(j==(Rows - 1))
				printf("\n");
		}
	}


	//*************************************************************************
	
	hipEventCreate(&start); 
	hipEventCreate(&stop);
	//************** Major/Minor Axes Length calculations Kernel ************** 
	dim3 dimGrid_2(BLOCKS_X_SCAN, BLOCKS_Y_SCAN);								
	dim3 dimBlock_2(Threads_x_Scan, Threads_y_Scan);	

	hipEventRecord( start, 0 );
	maj_min_scan_G <<<dimGrid_2, dimBlock_2>>>
		(
			Image_Out_dev, 
			dev_hor_scan, 
			dev_ver_scan, 
			dev_hor_max_obj, 
			dev_ver_max_obj
		);

	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time2, start, stop ); 

	printf("\nTIME to Run Maj/Min Axes Kernel = %f ms\n\n", time2);
	hipEventDestroy( start ); 
	hipEventDestroy( stop );

	hipMemcpy( hor_max_obj, dev_hor_max_obj, hor_max_obj_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( ver_max_obj, dev_ver_max_obj, ver_max_obj_size , hipMemcpyDeviceToHost ); 

	printf("\n Horizontal Max Values \n");
	printf(" ---------------------\n\n");
	for (i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for (j=0;j<BLOCKS_X_SCAN;j++)
			printf("%d ", hor_max_obj[i][j]);

		printf("\n"); 
	}

	printf("\n Vertical Max Values \n");
	printf(" ---------------------\n\n");
	for (i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for (j=0;j<BLOCKS_X_SCAN;j++)
			printf("%d ", ver_max_obj[i][j]);

		printf("\n"); 
	}

	//*************************************************************************

	hipEventCreate(&start); 
	hipEventCreate(&stop);
	//******************* Eccentricity calculations Kernel ******************** 
	dim3 dimGrid_3(1, 1);								
	dim3 dimBlock_3(2, 2, 1);	

	hipEventRecord( start, 0 );
	e_calc_G <<<dimGrid_3,dimBlock_3>>> 
		(
			dev_hor_max_obj, 
			dev_ver_max_obj, 
			dev_e_obj
		);

	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time3, start, stop ); 

	printf("\nTIME to Run Eccentricity Calculation Kernel = %f ms\n\n", time3);
	hipEventDestroy( start ); 
	hipEventDestroy( stop );

	hipMemcpy( e_obj, dev_e_obj, e_obj_size , hipMemcpyDeviceToHost ); 

	printf("\n"); 
	for(i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for(j=0;j<BLOCKS_X_SCAN;j++)
		{
			printf ("e[%d][%d] = %f\n",i,j,e_obj[i][j]);
		}
	}

	//*************************************************************************


	system("pause");
	return 0;
}
