//GPU implementation of Major Minor Axes Scan
//Author: Hasan Baig
//Email: hasan.baig@hotmail.com 
//Web: http://www.hasanbaig.com 

#include "iostream"
#include "conio.h" 
#include "time.h"
#include "../defines.h"
#include "../my_G_functions_SM.h"
#include "C:/ProgramData/NVIDIA Corporation/NVIDIA GPU Computing SDK 4.0/C/common/inc/cutil.h"




int main (void)
{
	int *dev_Image, 
		*dev_hor_scan, 
		*dev_ver_scan, 
		*dev_hor_max_obj, 
		*dev_ver_max_obj;
	
	hipEvent_t start, stop; 
	float time,time2;

	//unsigned int timer = 0;
	//cutCreateTimer( &timer );


	hipEventCreate(&start); 
	hipEventCreate(&stop);

	printf("\nMatrix Image \n\n");
	for (i=0;i<(TILE_WIDTH_SCAN*BLOCKS_Y_SCAN);i++)
	{
		for (j=0;j<WIDTH;j++)
		{
			printf("%d", Image[i][j]);
		}
		printf("\n"); 
	}

	dim3 dimGrid_1(BLOCKS_X_SCAN, BLOCKS_Y_SCAN);								
	dim3 dimBlock_1(Threads_x_Scan, Threads_y_Scan);	


	hipMalloc( (void**)&dev_Image, Image_size );
	hipMalloc( (void**)&dev_hor_scan, hor_scan_size);
	hipMalloc( (void**)&dev_ver_scan, ver_scan_size );
	hipMalloc( (void**)&dev_hor_max_obj, hor_max_obj_size );
	hipMalloc( (void**)&dev_ver_max_obj, ver_max_obj_size );

	hipMemcpy( dev_Image, Image, Image_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_hor_scan, hor_scan, hor_scan_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_ver_scan, ver_scan, ver_scan_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_hor_max_obj, hor_max_obj, hor_max_obj_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_ver_max_obj, ver_max_obj, ver_max_obj_size,hipMemcpyHostToDevice );

	//hipDeviceSynchronize();
	hipEventRecord( start, 0 );

	for(m=0;m<Loop_G;m++)
	{
	maj_min_scan_G_SM<<<dimGrid_1, dimBlock_1>>>(dev_Image, dev_hor_max_obj, dev_ver_max_obj);
	//maj_min_scan_G_SM_1<<<dimGrid_1, dimBlock_1>>>(dev_Image, dev_hor_scan, dev_ver_scan);
	}
	
	//hipDeviceSynchronize();
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop ); 
	hipEventElapsedTime( &time, start, stop ); 

	printf("\nTIME to Run Maj/Min_1 Axes Kernel = %f ms\n\n", time);
	//printf("\nTIME = %f ms\n\n", cutGetTimerValue( timer ));
	hipEventDestroy( start ); 
	hipEventDestroy( stop );

	hipMemcpy( hor_scan, dev_hor_scan, hor_scan_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( ver_scan, dev_ver_scan, ver_scan_size , hipMemcpyDeviceToHost ); 

/*
	//*************** For debugging ****************
	printf("\n Horizontal Scan \n");
	printf(" ---------------\n\n");
	for (i=0;i<(TILE_WIDTH_SCAN*BLOCKS_Y_SCAN);i++)
	{ 
		for (j=0;j<BLOCKS_X_SCAN;j++)
			printf("%d ", hor_scan[i][j]);
		
		printf("\n"); 
	}

	printf("\n Vertical Scan \n");
	printf(" -------------\n\n");
	for (i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for (j=0;j<WIDTH;j++)
			printf("%d ", ver_scan[i][j]);

		printf("\n"); 
	}
	


	//______----------------------------------------_______
	hipEventCreate(&start); 
	hipEventCreate(&stop);
	
	dim3 dimGrid_2(BLOCKS_X_SCAN, BLOCKS_Y_SCAN);								
	dim3 dimBlock_2(Threads_x_Scan, Threads_y_Scan);	

	hipEventRecord( start, 0 );
	maj_min_scan_G_SM_2<<<dimGrid_2, dimBlock_2>>>(dev_hor_scan, dev_ver_scan, dev_hor_max_obj, dev_ver_max_obj);
	
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time2, start, stop ); 

	printf("\nTIME to Run Maj/Min_2 Axes Kernel = %f ms\n\n", time2);
	hipEventDestroy( start ); 
	hipEventDestroy( stop );
*/



	hipMemcpy( hor_max_obj, dev_hor_max_obj, hor_max_obj_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( ver_max_obj, dev_ver_max_obj, ver_max_obj_size , hipMemcpyDeviceToHost ); 


	printf("\n Horizontal Max Values \n");
	printf(" ---------------------\n\n");
	for (i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for (j=0;j<BLOCKS_X_SCAN;j++)
			printf("%d ", hor_max_obj[i][j]);

		printf("\n"); 
	}

	printf("\n Vertical Max Values \n");
	printf(" ---------------------\n\n");
	for (i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for (j=0;j<BLOCKS_X_SCAN;j++)
			printf("%d ", ver_max_obj[i][j]);

		printf("\n"); 
	}
	
	//**********************************************
	system ("pause");
	return 0;

}


