//GPU implementation of Major Minor Axes Scan
#include "iostream"
#include "conio.h" 
#include "time.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/defines.h"
#include "D:/hasan_cuda_VC/CUDA_Stuff/my_G_functions.h"
#include "C:/ProgramData/NVIDIA Corporation/NVIDIA GPU Computing SDK 4.0/C/common/inc/cutil.h"



int main (void)
{
	int *dev_Image, *dev_hor_scan, *dev_ver_scan, *dev_hor_max_obj, *dev_ver_max_obj;
	
	hipEvent_t start, stop; 
	float time;

	unsigned int timer = 0;
	cutCreateTimer( &timer );


	hipEventCreate(&start); 
	hipEventCreate(&stop);

	printf("\nMatrix Image \n\n");
	for (i=0;i<(TILE_WIDTH_SCAN*BLOCKS_Y_SCAN);i++)
	{
		for (j=0;j<WIDTH;j++)
		{
			printf("%d", Image[i][j]);
		}
		printf("\n"); 
	}

	dim3 dimGrid(BLOCKS_X_SCAN, BLOCKS_Y_SCAN);								
	dim3 dimBlock(Threads_x_Scan, Threads_y_Scan);	


	hipMalloc( (void**)&dev_Image, Image_size );
	hipMalloc( (void**)&dev_hor_scan, hor_scan_size);
	hipMalloc( (void**)&dev_ver_scan, ver_scan_size );
	hipMalloc( (void**)&dev_hor_max_obj, hor_max_obj_size );
	hipMalloc( (void**)&dev_ver_max_obj, ver_max_obj_size );

	hipMemcpy( dev_Image, Image, Image_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_hor_max_obj, hor_max_obj, hor_max_obj_size,hipMemcpyHostToDevice );
	hipMemcpy( dev_ver_max_obj, ver_max_obj, ver_max_obj_size,hipMemcpyHostToDevice );
	
	//hipDeviceSynchronize();
	hipEventRecord( start, 0 );

	for(m=0;m<Loop_G;m++)
	maj_min_scan_G<<<dimGrid, dimBlock>>>(dev_Image, dev_hor_scan, dev_ver_scan, dev_hor_max_obj, dev_ver_max_obj);
	
	//hipDeviceSynchronize();
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop ); 
	hipEventElapsedTime( &time, start, stop ); 

	printf("\nTIME = %f ms\n\n", time);
	//printf("\nTIME = %f ms\n\n", cutGetTimerValue( timer ));
	hipEventDestroy( start ); 
	hipEventDestroy( stop );

	hipMemcpy( hor_scan, dev_hor_scan, hor_scan_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( ver_scan, dev_ver_scan, ver_scan_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( hor_max_obj, dev_hor_max_obj, hor_max_obj_size , hipMemcpyDeviceToHost ); 
	hipMemcpy( ver_max_obj, dev_ver_max_obj, ver_max_obj_size , hipMemcpyDeviceToHost ); 

	//*************** For debugging ****************
	printf("\n Horizontal Scan \n");
	printf(" ---------------\n\n");
	for (i=0;i<(TILE_WIDTH_SCAN*BLOCKS_Y_SCAN);i++)
	{ 
		for (j=0;j<BLOCKS_X_SCAN;j++)
			printf("%d ", hor_scan[i][j]);
		
		printf("\n"); 
	}

	printf("\n Vertical Scan \n");
	printf(" -------------\n\n");
	for (i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for (j=0;j<WIDTH;j++)
			printf("%d ", ver_scan[i][j]);

		printf("\n"); 
	}

	printf("\n Horizontal Axes Max Values \n");
	printf(" ---------------------\n\n");
	for (i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for (j=0;j<BLOCKS_X_SCAN;j++)
			printf("Obj[%d]=%d ",i+j, hor_max_obj[i][j]);

		printf("\n"); 
	}

	printf("\n Vertical Axes Max Values \n");
	printf(" ---------------------\n\n");
	for (i=0;i<BLOCKS_Y_SCAN;i++)
	{
		for (j=0;j<BLOCKS_X_SCAN;j++)
			printf("Obj[%d]=%d ",i+j, ver_max_obj[i][j]);

		printf("\n"); 
	}
	
	//**********************************************
	system ("pause");
	return 0;
}


